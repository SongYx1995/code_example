
#include <hip/hip_runtime.h>
#include<iostream>
#define HANDLE_ERROR(ret) \
    {\
        if(ret != hipSuccess)\
            std::cerr<<"cuda wrong"<<std::endl;\
    }
__global__ void add(int* a,int* b,int* c){
    int idx=threadIdx.x;
    c[idx]=a[idx]+b[idx];
}
int main(){
    int a = 123;
    int b= 234;
    int c;
    int *dev_a,*dev_b,*dev_c;
    HANDLE_ERROR(hipMalloc(&dev_a,sizeof(int)));
    HANDLE_ERROR(hipMalloc(&dev_b,sizeof(int)));
    HANDLE_ERROR(hipMalloc(&dev_c,sizeof(int)));

    HANDLE_ERROR(hipMemcpy(dev_a,&a,sizeof(int),hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b,&b,sizeof(int),hipMemcpyHostToDevice));

    add<<<1,3,0>>>(dev_a,dev_b,dev_c);
    
    HANDLE_ERROR(hipMemcpy(&c,dev_c,sizeof(int),hipMemcpyDeviceToHost));

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    std::cout<<c;
}